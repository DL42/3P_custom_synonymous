#include "hip/hip_runtime.h"
/*
 * run.cu
 *
 *      Author: David Lawrie
 */

#include <fstream>
#include "go_fish.cuh"
#include "spectrum.h"
#include "run.h"

void run_speed_test()
{
	//----- warm up scenario parameters -----
	GO_Fish::allele_trajectories a;
	a.sim_input_constants.num_generations = pow(10.f,5);//36;//50;//
	a.sim_input_constants.num_sites = 2*pow(10.f,7); //number of sites
	a.sim_input_constants.num_populations = 1; //number of populations
	a.sim_input_constants.seed1 = 0xbeeff00d; //random number seeds
	a.sim_input_constants.seed2 = 0xdecafbad;
	Sim_Model::F_mu_h_constant mutation(pow(10.f,-9)); //per-site mutation rate
	Sim_Model::F_mu_h_constant inbreeding(1.f); //constant inbreeding
	Sim_Model::demography_constant demography(pow(10.f,5)*(1+inbreeding(0,0))); //number of individuals in population, set to maintain consistent effective number of chromosomes
	Sim_Model::migration_constant_equal migration(0.f,a.sim_input_constants.num_populations); //constant migration rate
	float gamma = 0; //effective selection
	Sim_Model::selection_constant selection(gamma/(2*demography(0,0))); //constant selection coefficient
	Sim_Model::F_mu_h_constant dominance(0.f); //constant allele dominance
	Sim_Model::bool_off preserve; //don't preserve alleles from any generation
	Sim_Model::bool_off sample_strategy; //only sample final generation
	//----- end warm up scenario parameters -----

	//----- warm up GPU -----
	GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,preserve,sample_strategy);
	std::cout<<std::endl<<"final number of mutations: " << a.maximal_num_mutations() << std::endl;

	//----- print allele counts x to x+y of warm up GPU scenario -----
	bool printSFS = true; //calculate and print out the SFS
	int start_index = 0;
	int print_num = 50;
	Spectrum::SFS mySFS;
	if(printSFS){
		Spectrum::population_frequency_histogram(mySFS,a,0,0);
		std::cout<< "allele count\t# mutations"<< std::endl;
		for(int printIndex = start_index; printIndex < min((mySFS.sample_size[0]-start_index),start_index+print_num); printIndex++){ std::cout<< (printIndex) << "\t" << mySFS.frequency_spectrum[printIndex] << std::endl;}
	}
	//----- end print allele counts x to x+y of warm up GPU scenario -----

	//GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy);
	GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,preserve,sample_strategy);
	//----- end warm up GPU -----

	//----- speed test scenario parameters -----
    hipEvent_t start, stop;
    float elapsedTime;
    int num_iter = 10;
    a.sim_input_constants.compact_interval = 35;
    a.sim_input_constants.num_generations = pow(10.f,3);
    a.sim_input_constants.num_sites = 2*pow(10.f,7);
    a.sim_input_constants.seed1 = 0xbeeff00d; //random number seeds
    a.sim_input_constants.seed2 = 0xdecafbad;
	//----- end speed test scenario parameters -----

    //----- speed test -----
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for(int i = 0; i < num_iter; i++){ GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,Sim_Model::bool_off(),Sim_Model::bool_off()); }

	elapsedTime = 0;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout<< std::endl<<"final number of mutations: " << a.maximal_num_mutations() << std::endl;
	std::cout<< std::endl<<a.num_time_samples()<<std::endl;
	for(int i = 0; i < a.num_time_samples(); i++){
		std::cout<<a.mutation_ID(1).toString()<<" "<<a.frequency(i,0,1)<<"\t"<<a.mutation_ID(50000).toString()<<" "<<a.frequency(i,0,50000)<<"\t"<<a.mutation_ID(100000).toString()<<" "<<a.frequency(i,0,100000)<<std::endl;
	}

	std::cout<<std::endl;

	printf("time elapsed: %f\n\n", elapsedTime/num_iter);
	//----- end speed test -----
	//
}

////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////

void run_prev_sim_n_allele_traj_test(){
	GO_Fish::allele_trajectories a;
	a.sim_input_constants.num_generations = 5*pow(10.f,4);//36;//50;//
	a.sim_input_constants.num_sites = 2*pow(10.f,7); //number of sites
	a.sim_input_constants.num_populations = 1; //number of populations
	a.sim_input_constants.seed1 = 0xbeeff00d; //random number seeds
	a.sim_input_constants.seed2 = 0xdecafbad;
	a.sim_input_constants.init_mse = false;
	Sim_Model::F_mu_h_constant mutation1(1.07*pow(10.f,-9)); //per-site mutation rate
	Sim_Model::F_mu_h_constant inbreeding(1.f); //constant inbreeding
	Sim_Model::demography_constant demography(pow(10.f,4)*(1+inbreeding(0,0))); //number of individuals in population, set to maintain consistent effective number of chromosomes
	Sim_Model::migration_constant_equal migration(0.f,a.sim_input_constants.num_populations); //constant migration rate
	float gamma = 0; //effective selection
	Sim_Model::selection_constant selection(gamma/(2*demography(0,0))); //constant selection coefficient
	Sim_Model::F_mu_h_constant dominance(0.f); //constant allele dominance
	Sim_Model::bool_off dont_preserve; //don't preserve alleles
	Sim_Model::bool_off dont_sample; //don't sample alleles
	Sim_Model::bool_on sample; //sample alleles
	Sim_Model::bool_pulse<Sim_Model::bool_off,Sim_Model::bool_on> sample_strategy(dont_sample,sample,0,a.sim_input_constants.num_generations); //sample starting generation of second simulation


	GO_Fish::run_sim(a,mutation1,demography,migration,selection,inbreeding,dominance,dont_preserve,dont_sample); //only sample final generation
	std::cout<<std::endl<<"final number of mutations: " << a.maximal_num_mutations() << std::endl;

	GO_Fish::allele_trajectories c(a);

	a.sim_input_constants.num_generations = pow(10.f,3);//36;//50;//
	a.sim_input_constants.prev_sim_sample = 0;
	Sim_Model::F_mu_h_constant mutation2(pow(10.f,-9)); //per-site mutation rate
	GO_Fish::run_sim(a,mutation2,demography,migration,selection,inbreeding,dominance,dont_preserve,sample_strategy,c);

	std::cout<<std::endl<<"number of time samples: " << a.num_time_samples();
	std::cout<<std::endl<<"starting number of mutations: " << a.num_mutations_time_sample(0) <<std::endl<<"final number of mutations: " << a.maximal_num_mutations() << std::endl;
	int mutation_range_begin = 0; int mutation_range_end = 10;
	std::cout<<"mutation IDs\tstart gen "<<a.sampled_generation(0)<<"\tfrequency\tfinal gen "<<a.final_generation()<<"\tfrequency"<<std::endl;
	for(int i = mutation_range_begin; i < mutation_range_end; i++){ std::cout<<"\t\t"<<a.mutation_ID(i)<<"\t"<<a.frequency(0,0,i)<<"\t\t"<<a.mutation_ID(i)<<"\t"<<a.frequency(1,0,i)<<std::endl; }
	mutation_range_begin = 11000; mutation_range_end = 11010;
	std::cout<<"mutation IDs\tID\tstart_frequency\tfinal_frequency"<<std::endl;
	for(int i = mutation_range_begin; i < mutation_range_end; i++){ std::cout<<"\t\t"<<a.mutation_ID(i)<<"\t"<<a.frequency(0,0,i)<<"\t"<<a.frequency(1,0,i)<<std::endl; }

	GO_Fish::allele_trajectories b = a; //tests both copy-constructor and copy-assignment

	std::cout<<std::endl<<"number of time samples: " << b.num_time_samples();
	std::cout<<std::endl<<"starting number of mutations: " << b.num_mutations_time_sample(0) <<std::endl<<"final number of mutations: " << b.maximal_num_mutations() << std::endl;
	mutation_range_begin = 0; mutation_range_end = 10;
	std::cout<<"mutation IDs\tstart gen "<<b.sampled_generation(0)<<"\tfrequency\tfinal gen "<<b.final_generation()<<"\tfrequency"<<std::endl;
	for(int i = mutation_range_begin; i < mutation_range_end; i++){ std::cout<<"\t\t"<<b.mutation_ID(i)<<"\t"<<b.frequency(0,0,i)<<"\t\t"<<b.mutation_ID(i)<<"\t"<<b.frequency(1,0,i)<<std::endl; }
	mutation_range_begin = 11000; mutation_range_end = 11010;
	std::cout<<"mutation IDs\tID\tstart_frequency\tfinal_frequency"<<std::endl;
	for(int i = mutation_range_begin; i < mutation_range_end; i++){ std::cout<<"\t\t"<<b.mutation_ID(i)<<"\t"<<b.frequency(0,0,i)<<"\t"<<b.frequency(1,0,i)<<std::endl; }

	std::ofstream outfile;
	outfile.open("afile.dat");
	outfile<<a;
	outfile.close();
	outfile.open("bfile.dat");
	outfile<<b;
	outfile.close();

	a.sim_input_constants.init_mse = true;
	a.sim_input_constants.seed1 = 0xdecafbad; //random number seeds
	a.sim_input_constants.seed2 = 0xbeeff00d;
	a.sim_input_constants.num_generations = pow(10.f,3);//36;//50;//
	GO_Fish::run_sim(a,mutation2,demography,migration,selection,inbreeding,dominance,dont_preserve,dont_sample);
	std::cout<<std::endl<<"final number of mutations: " << a.maximal_num_mutations() << std::endl;
}

////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////

double gx(double x, double gamma, double mu_site, double L){
	if(gamma != 0) return 2*mu_site*L*(1-exp(-1*gamma*(1-x)))/((1-exp(-1*gamma))*x*(1-x));
	return 2*mu_site*L/x;
}


double* G(double gamma,double mu_site, double L, double N_chrome){
	double total_SNPs = 0;
	double* g = new double[(int)N_chrome];

	for(int j = 1; j <= (N_chrome - 1); j++){
		double freq = j/(N_chrome);
		g[j] = gx(freq, gamma, mu_site, L);
		total_SNPs += g[j];
	}

	g[0] = L-total_SNPs;

	return g;
}

void run_validation_test(){

	GO_Fish::allele_trajectories b;
	float h = 0.5; //dominance
	float F = 0.0; //inbreeding
	int N_ind = pow(10.f,5);//300;// //bug at N_ind = 300, F =0.0/1.0, gamma = 0//number of individuals in population, set to maintain consistent effective number of chromosomes across all inbreeding coefficients
    float gamma = 0*(1+F); //effective selection //set to maintain consistent level of selection across all inbreeding coefficients for the same effective number of chromosomes, drift and selection are invariant with respect to inbreeding
	float mu = pow(10.f,-9); //per-site mutation rate
	int total_number_of_generations = pow(10.f,3);//0;//1000;//1;//36;//
	b.sim_input_constants.num_generations = total_number_of_generations;
	b.sim_input_constants.num_sites = 20*2*pow(10.f,7); //number of sites
	float m = 0.00; //migration rate
	b.sim_input_constants.num_populations = 1; //number of populations
	int num_iter = 50;
    b.sim_input_constants.compact_interval = 20;
   // double* expectation = G(gamma,mu, b.sim_input_constants.num_sites, 2.0*N_ind/(1.0+F));
    //double expected_total_SNPs = b.sim_input_constants.num_sites-expectation[0];
    Spectrum::SFS * my_spectra = new Spectrum::SFS[num_iter];

    hipEvent_t start, stop;
    float elapsedTime;
    int sample_size = 200;
	for(int i = 0; i < num_iter; i++){
		if(i == round(num_iter/2.f)){
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start, 0);
		}

		b.sim_input_constants.seed1 = 0xbeeff00d + 2*i; //random number seeds
		b.sim_input_constants.seed2 = 0xdecafbad - 2*i;
		GO_Fish::run_sim((b), Sim_Model::F_mu_h_constant(mu), Sim_Model::demography_constant(N_ind), Sim_Model::migration_constant_equal(m,b.sim_input_constants.num_populations), Sim_Model::selection_constant(gamma,Sim_Model::demography_constant(N_ind),Sim_Model::F_mu_h_constant(F)), Sim_Model::F_mu_h_constant(F), Sim_Model::F_mu_h_constant(h), Sim_Model::bool_off(), Sim_Model::bool_off());
		Spectrum::site_frequency_spectrum(my_spectra[i],(b),0,0,sample_size);
		//if(i==0){ std::cout<< "dispersion/chi-gram of number of mutations:"<<std::endl; }
		//std::cout<<b.maximal_num_mutations()<<std::endl;
		//std::cout<< (int)expected_total_SNPs << "\t" << b.maximal_num_mutations() << "\t" << ((b.maximal_num_mutations() - expected_total_SNPs)/expected_total_SNPs) << std::endl;
	}

	elapsedTime = 0;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout<<"\ntime elapsed: "<< 2*elapsedTime/num_iter<<std::endl;
	//----- end speed test -----
	//
	//if(my_spectra[0].frequency_spectrum[0] < 0){ std::cout<<std::endl<<0<<"\t"<<my_spectra[0].frequency_spectrum[0]<<std::endl; }
	std::cout<<std::endl<<"SFS :"<<std::endl<< "allele count\tavg# mutations\tstandard dev\tcoeff of variation (aka relative standard deviation)"<< std::endl;
	float avg_num_mutations = 0;
	for(int i = 1; i < sample_size; i++){
		double avg = 0;
		double std = 0;
		float num_mutations;
		for(int j = 0; j < num_iter; j++){ num_mutations = my_spectra[j].num_mutations; avg += my_spectra[j].frequency_spectrum[i]/(num_iter*num_mutations); if(i==1){ avg_num_mutations += ((float)num_mutations)/num_iter; }}
		for(int j = 0; j < num_iter; j++){ num_mutations = my_spectra[j].num_mutations; std += 1.0/(num_iter-1)*pow(my_spectra[j].frequency_spectrum[i]/num_mutations-avg,2); }
		std = sqrt(std);
		std::cout<<i<<"\t"<<avg<<"\t"<<std<<"\t"<<(std/avg)<<std::endl;
		//std::cout<<avg<<std::endl;
	}
	std::cout<<avg_num_mutations<<std::endl;
	//delete [] expectation;
	delete [] my_spectra;
}
