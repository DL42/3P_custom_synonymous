/*
 * go_fish_data_struct.cu
 *
 *      Author: David Lawrie
 *      GO Fish data structures
 */

#include "../include/go_fish_data_struct.h"
#include "../source/shared.cuh"

namespace GO_Fish{

allele_trajectories::time_sample::time_sample(): num_populations(0), num_mutations(0), num_sites(0), sampled_generation(0) { mutations_freq = 0; mutations_ID = 0; extinct = 0; Nchrom_e = 0; /*set pointers to NULL*/}
allele_trajectories::time_sample::~time_sample(){
	if(mutations_freq){ cudaCheckErrors(hipHostFree(mutations_freq),-1,-1); mutations_freq = NULL; }
	if(mutations_ID){ cudaCheckErrors(hipHostFree(mutations_ID),-1,-1); mutations_ID = NULL; }
	if(extinct){ delete [] extinct; extinct = NULL; }
	if(Nchrom_e){ delete [] Nchrom_e; Nchrom_e = NULL; }
}

allele_trajectories::allele_trajectories(): length(0) { time_samples = 0; }
allele_trajectories::~allele_trajectories(){ free_memory(); }
allele_trajectories::sim_input_constants::sim_input_constants(): seed1(0xbeeff00d), seed2(0xdecafbad), num_generations(0), num_sites(1000), num_discrete_DFE_categories(1), num_populations(1), init_mse(true), prev_sim_sample(-1), compact_interval(35), device(-1) {}

}/* ----- end namespace GO_Fish ----- */
