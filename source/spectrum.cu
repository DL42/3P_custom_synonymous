#include "hip/hip_runtime.h"
/*
 * spectrum.cu
 *
 *      Author: David Lawrie
 */

#include "../include/spectrum.h"
#include "../source/shared.cuh"
#include <cub/device/device_scan.cuh>

namespace SPECTRUM{

class transfer_allele_trajectories{

	struct time_sample{
		float * mutations_freq; //allele frequency of mutations in final generation
		GO_Fish::mutID * mutations_ID; //unique ID consisting of generation, population, threadID, and device
		bool * extinct; //extinct[pop] == true, flag if population is extinct by end of simulation
		int * Nchrom_e; //effective number of chromosomes in each population
		int num_populations; //number of populations in freq array (array length, rows)
		int num_mutations; //number of mutations in array (array length for age/freq, columns)
		float num_sites; //number of sites in simulation
		int sampled_generation; //number of generations in the simulation at point of sampling

		time_sample(): num_populations(0), num_mutations(0), num_sites(0), sampled_generation(0) { mutations_freq = 0; mutations_ID = 0; extinct = 0; Nchrom_e = 0; }
		time_sample(const GO_Fish::allele_trajectories & in, int sample_index): num_populations(in.time_samples[sample_index]->num_populations), num_mutations(in.time_samples[sample_index]->num_mutations), num_sites(in.time_samples[sample_index]->num_sites), sampled_generation(in.time_samples[sample_index]->sampled_generation){
			mutations_freq = in.time_samples[sample_index]->mutations_freq;
			mutations_ID = in.time_samples[sample_index]->mutations_ID;
			extinct = in.time_samples[sample_index]->extinct;
			Nchrom_e = in.time_samples[sample_index]->Nchrom_e;
		}
		~time_sample(){ mutations_freq = 0; mutations_ID = 0; extinct = 0; Nchrom_e = 0; } //don't actually delete information, just null pointers as this just points to the real data held
	};

	time_sample ** time_samples;
	unsigned int length;

	//----- initialization parameters -----
	struct sim_input_constants{
		int seed1;
		int seed2;
		int num_generations;
		float num_sites;
		int num_discrete_DFE_categories;
		int num_populations;
		bool init_mse;
		int prev_sim_sample;
		int compact_rate;
		int device;

		sim_input_constants();
		sim_input_constants(const GO_Fish::allele_trajectories & in){
			seed1 = in.sim_input_constants.seed1;
			seed2 = in.sim_input_constants.seed2;
			num_generations = in.sim_input_constants.num_generations;
			num_sites = in.sim_input_constants.num_sites;
			num_discrete_DFE_categories = in.sim_input_constants.num_discrete_DFE_categories;
			num_populations = in.sim_input_constants.num_populations;
			init_mse = in.sim_input_constants.init_mse;
			prev_sim_sample = in.sim_input_constants.prev_sim_sample;
			compact_rate = in.sim_input_constants.compact_rate;
			device = in.sim_input_constants.device;
		}
	}sim_input_constants;
	//----- end -----

public:

	transfer_allele_trajectories(): length(0) { time_samples = 0; }

	transfer_allele_trajectories(const GO_Fish::allele_trajectories & in): sim_input_constants(in){
		if(!in.time_samples || in.length == 0){ fprintf(stderr,"error transferring allele_trajectories to spectrum: empty allele_trajectories\n"); exit(1); }
		length = in.length;
		time_samples = new time_sample *[length];

		for(int i = 0; i < length; i++){ time_samples[i] = new time_sample(in,i); }
	}

	friend sfs site_frequency_spectrum(const GO_Fish::allele_trajectories & all_results, const int sample_index, const int population_index, const unsigned int sample_size, int cuda_device);

	~transfer_allele_trajectories(){ time_samples = 0; length = 0; } //don't actually delete anything, this is just a pointer class, actual data held by GO_Fish::trajectory
};

__global__ void population_hist(unsigned int * out_histogram, float * in_mutation_freq, int Nchrome_e, int num_mutations, int num_sites){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;

	for(int id = myID; id < num_mutations; id+= blockDim.x*gridDim.x){
		int index = round(Nchrome_e*in_mutation_freq[id]);
		atomicAdd(&out_histogram[index],1);
	}
	if(myID == 0){  out_histogram[0] = num_sites - num_mutations;  }
}

__global__ void uint_to_float(float * out_array, unsigned int * in_array, int N){
	int myID =  blockIdx.x*blockDim.x + threadIdx.x;
	for(int id = myID; id < N; id+= blockDim.x*gridDim.x){ out_array[id] = in_array[id]; }
}

/*    function [P_samp] = sample_prob(Npop,Nsamp)
        x = (0:(2*Npop-1))/(2*Npop);
        P_samp = zeros((Nsamp+1),length(x));
        for k = 0:Nsamp
            P_samp((k+1),:) = binopdf(k,Nsamp,x);
        end
    end

    function [prob] = prob(k,rho,P_samp)
        if(length(P_samp) > length(rho))
            prob = dot(rho,P_samp((k+1),(2:length(P_samp))));
        else
            prob = dot(rho,P_samp((k+1),:));
        end

    end*/

//pdivq = p/q
__global__ void binom_fract(float * binom_fract, float q, float pdivq, int half_n, int n){
	int myIDx =  blockIdx.x*blockDim.x + threadIdx.x;

	for(int idx = (myIDx+1); idx < half_n; idx+= blockDim.x*gridDim.x){ binom_fract[idx] =  ((n+1.f-idx)/((float)idx))*pdivq; }
	if(myIDx == 0){ binom_fract[0] = pow(q,n); }
}

struct CustomMultiply
{
    template <typename T>
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { return a*b; }
};

__global__ void print_Device_array_float(float * array, int num){

		//if(i%1000 == 0){ printf("\n"); }
	for(int j = 0; j < num; j++){ printf("%d: %f\t",j,array[j]); }
	printf("\n");
}


sfs::sfs(): num_populations(0), num_sites(0), sampled_generation(0) {frequency_spectrum = NULL; populations = NULL; sample_size = NULL;}
sfs::~sfs(){ if(frequency_spectrum){ cudaCheckErrors(hipHostFree(frequency_spectrum),-1,-1); frequency_spectrum = NULL; } if(populations){ delete[] populations; populations = NULL; } if(sample_size){ delete[] sample_size; sample_size = NULL; }}

//single-population sfs
sfs site_frequency_spectrum(const GO_Fish::allele_trajectories & all_results, const int sample_index, const int population_index, const unsigned int sample_size, int cuda_device){

	set_cuda_device(cuda_device);

	hipStream_t stream;

	cudaCheckErrors(hipStreamCreate(&stream),-1,-1);

	float * d_mutations_freq;
	unsigned int * d_pop_histogram;
	float * d_histogram, * h_histogram;
	transfer_allele_trajectories sample(all_results);
	if(!(sample_index >= 0 && sample_index < sample.length) || !(population_index >= 0 && population_index < sample.sim_input_constants.num_populations)){
		fprintf(stderr,"site_frequency_spectrum error: requested indices out of bounds: sample %d\t[0 %d)\tpopulation %d\t[0 %d)\n",sample_index,sample.length,population_index,sample.sim_input_constants.num_populations); exit(1);
	}

	int num_levels = sample_size;
	int population_size = sample.time_samples[sample_index]->Nchrom_e[population_index];
	if(sample_size == 0){ num_levels = population_size; }

	cudaCheckErrorsAsync(hipMalloc((void**)&d_mutations_freq, sample.time_samples[sample_index]->num_mutations*sizeof(float)),-1,-1);
	cudaCheckErrorsAsync(hipMalloc((void**)&d_pop_histogram, num_levels*sizeof(unsigned int)),-1,-1);
	cudaCheckErrorsAsync(hipMalloc((void**)&d_histogram, num_levels*sizeof(float)),-1,-1);
	cudaCheckErrorsAsync(hipMemsetAsync(d_pop_histogram, 0, num_levels*sizeof(unsigned int), stream),-1,-1);
	cudaCheckErrorsAsync(hipMemcpyAsync(d_mutations_freq, &sample.time_samples[sample_index]->mutations_freq[population_index*sample.time_samples[sample_index]->num_mutations], sample.time_samples[sample_index]->num_mutations*sizeof(float), hipMemcpyHostToDevice, stream),-1,-1);

	population_hist<<<50,1024,0,stream>>>(d_pop_histogram, d_mutations_freq, population_size, sample.time_samples[sample_index]->num_mutations, sample.time_samples[sample_index]->num_sites);
	cudaCheckErrorsAsync(hipPeekAtLastError(),-1,-1);
	/*         for k = 1:(N_samp-1)
	            mix_prob = prob(k,rho_mix,P_samp);
	            %G(k) = round(mix_prob*m_mix);
	            G(k) = mix_prob*m_mix;
	            m_samp = m_samp + G(k);
	        end   */

	int num_threads = 1024;
	int num_blocks = max(num_levels/num_threads,1);
	if(sample_size == 0){
		uint_to_float<<<num_blocks,num_threads,0,stream>>>(d_histogram, d_pop_histogram, num_levels);
		cudaCheckErrorsAsync(hipPeekAtLastError(),-1,-1);
	}
	//else{
		int temp = num_levels;
		num_levels = 1000;
		num_threads = 128;
		num_blocks = min(num_levels/num_threads,1);
		int half_n;
		if(num_levels % 2 == 0){ half_n = num_levels/2+1; }
		else{ half_n = (num_levels+1)/2; }

		float * d_binom_fract;
		cudaCheckErrorsAsync(hipMalloc((void**)&d_binom_fract, half_n*sizeof(float)),-1,-1);
		const dim3 gridsize(10,10,1);
		float p = (20000.f)/(float(population_size));
		float q = (population_size - 20000.f)/(float(population_size));
		p = 0.01;
		q = 0.99;
		//printf("p: ");
		float pdivq = (double)p/(double)q;
		binom_fract<<<50,128,0,stream>>>(d_binom_fract, q, pdivq, half_n, num_levels);
		cudaCheckErrorsAsync(hipPeekAtLastError(),-1,-1);

		print_Device_array_float<<<1,1,0,stream>>>(d_binom_fract, 50);

		float * d_binom;
		CustomMultiply mul_op;
		cudaCheckErrorsAsync(hipMalloc((void**)&d_binom, half_n*sizeof(float)),-1,-1);

		void *d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_binom_fract, d_binom, mul_op, half_n);
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_binom_fract, d_binom, mul_op, half_n);
		cudaCheckErrorsAsync(hipFree(d_temp_storage),-1,-1);
		cudaCheckErrorsAsync(hipFree(d_binom_fract),-1,-1);

		print_Device_array_float<<<1,1,0,stream>>>(d_binom, 50);

		cudaCheckErrorsAsync(hipFree(d_binom),-1,-1);
		num_levels = temp;
	//}

	cudaCheckErrors(hipHostMalloc((void**)&h_histogram, num_levels*sizeof(float)),-1,-1);
	cudaCheckErrorsAsync(hipMemcpyAsync(h_histogram, d_histogram, num_levels*sizeof(int), hipMemcpyDeviceToHost, stream),-1,-1);

	if(hipStreamQuery(stream) != hipSuccess){ cudaCheckErrors(hipStreamSynchronize(stream), -1, -1); } //wait for writes to host to finish

	sfs mySFS;
	mySFS.frequency_spectrum = h_histogram;
	mySFS.num_populations = 1;
	mySFS.sample_size = new int[1];
	mySFS.sample_size[0] = num_levels;
	mySFS.num_sites = sample.time_samples[sample_index]->num_sites;
	mySFS.populations = new int[1];
	mySFS.populations[0] = population_index;
	mySFS.sampled_generation = sample.time_samples[sample_index]->sampled_generation;

	//cudaCheckErrorsAsync(hipFree(d_temp_storage),-1,-1);
	cudaCheckErrorsAsync(hipFree(d_mutations_freq),-1,-1);
	cudaCheckErrorsAsync(hipFree(d_histogram),-1,-1);
	cudaCheckErrorsAsync(hipStreamDestroy(stream),-1,-1)

	return mySFS;
}

} /*----- end namespace SPECTRUM ----- */
