#include "hip/hip_runtime.h"
/*
 * shared.cu
 *
 *      Author: David Lawrie
 *      for structures and functions used by both go_fish and by sfs
 */

#include "shared.cuh"

sim_result::sim_result(): num_populations(0), num_mutations(0), num_sites(0), total_generations(0) { mutations_freq = NULL; mutations_ID = NULL; extinct = NULL; }
sim_result::~sim_result(){ if(mutations_freq){ cudaCheckErrors(hipHostFree(mutations_freq),-1,-1); } if(mutations_ID){ cudaCheckErrors(hipHostFree(mutations_ID),-1,-1); } if(extinct){ delete [] extinct; } }

__device__ __noinline__ int Rand1(unsigned int i, float mean, float var, float N){
	if(mean <= 6){ return poiscdfinv(uint_float_01(i), mean); }
	else if(mean >= N-6){ return N - poiscdfinv(uint_float_01(i), N-mean); } //flip side of binomial, when 1-p is small
	return round(normcdfinv(uint_float_01(i))*sqrtf(var)+mean);
}
