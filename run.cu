#include "hip/hip_runtime.h"
/*
 * run.cu
 *
 *      Author: David Lawrie
 */

#include "go_fish.cuh"
#include "sfs.h"
#include "run.h"

void run_speed_test()
{
	//----- warm up scenario parameters -----
	GO_Fish::allele_trajectories a;
	a.sim_input_constants.num_generations = pow(10.f,5);//36;//50;//
	a.sim_input_constants.num_sites = 2*pow(10.f,7); //number of sites
	a.sim_input_constants.num_populations = 1; //number of populations
	a.sim_input_constants.seed1 = 0xbeeff00d; //random number seeds
	a.sim_input_constants.seed2 = 0xdecafbad;
	bool DFE = false;
	GO_Fish::const_parameter mutation(pow(10.f,-9)); //per-site mutation rate
	GO_Fish::const_parameter inbreeding(1.f); //constant inbreeding
	GO_Fish::const_demography demography(pow(10.f,5)*(1+inbreeding(0,0))); //number of individuals in population, set to maintain consistent effective number of chromosomes
	GO_Fish::const_equal_migration migration(0.f,a.sim_input_constants.num_populations); //constant migration rate
	float gamma = 0; //effective selection
	GO_Fish::const_selection selection(gamma/(2*demography(0,0))); //constant selection coefficient
	GO_Fish::const_parameter dominance(0.f); //constant allele dominance
	GO_Fish::do_nothing preserve; //don't preserve alleles from any generation
	GO_Fish::do_nothing sample_strategy; //only sample final generation
	//----- end warm up scenario parameters -----

	//----- warm up GPU -----
	bool printSFS = true; //calculate and print out the SFS
	GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy,GO_Fish::time_sample());
	std::cout<<std::endl<<"final number of mutations: " << a[0]->num_mutations << std::endl;

	//----- print allele counts x to x+y of warm up GPU scenario -----
	int start_index = 0;
	int print_num = 50;
	if(printSFS){
		SFS::sfs mySFS = SFS::site_frequency_spectrum(a[0],0);
		std::cout<< "allele count\t# mutations"<< std::endl;
		for(int printIndex = start_index; printIndex < min((mySFS.num_samples[0]-start_index),start_index+print_num); printIndex++){ std::cout<< (printIndex) << "\t" << mySFS.frequency_spectrum[printIndex] << std::endl;}
	}
	//----- end print allele counts x to x+y of warm up GPU scenario -----

	GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy,GO_Fish::time_sample());
	//----- end warm up GPU -----

	//----- speed test scenario parameters -----
    hipEvent_t start, stop;
    float elapsedTime;
    int num_iter = 10;
    a.sim_input_constants.compact_rate = 20;
    a.sim_input_constants.num_generations = pow(10.f,3);
    a.sim_input_constants.num_sites = 10*2*pow(10.f,7);
    a.sim_input_constants.seed1 = 0xbeeff00d; //random number seeds
    a.sim_input_constants.seed2 = 0xdecafbad;
	DFE = true;
	//----- end speed test scenario parameters -----

    //----- speed test -----
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for(int i = 0; i < num_iter; i++){
		GO_Fish::run_sim(a,mutation,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy,GO_Fish::time_sample());
		if(i==0){ std::cout<< std::endl<<"final number of mutations: " << a[0]->num_mutations << std::endl; }
	}

	elapsedTime = 0;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("time elapsed: %f\n\n", elapsedTime/num_iter);
	//----- end speed test -----
	//
	hipDeviceSynchronize();
	hipDeviceReset();
}

////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////

void run_prev_sim_test(){
	GO_Fish::allele_trajectories a;
	a.sim_input_constants.num_generations = 5*pow(10.f,4);//36;//50;//
	a.sim_input_constants.num_sites = 2*pow(10.f,7); //number of sites
	a.sim_input_constants.num_populations = 1; //number of populations
	a.sim_input_constants.seed1 = 0xbeeff00d; //random number seeds
	a.sim_input_constants.seed2 = 0xdecafbad;
	a.sim_input_constants.init_mse = false;
	bool DFE = false;
	GO_Fish::const_parameter mutation1(1.07*pow(10.f,-9)); //per-site mutation rate
	GO_Fish::const_parameter inbreeding(1.f); //constant inbreeding
	GO_Fish::const_demography demography(pow(10.f,4)*(1+inbreeding(0,0))); //number of individuals in population, set to maintain consistent effective number of chromosomes
	GO_Fish::const_equal_migration migration(0.f,a.sim_input_constants.num_populations); //constant migration rate
	float gamma = 0; //effective selection
	GO_Fish::const_selection selection(gamma/(2*demography(0,0))); //constant selection coefficient
	GO_Fish::const_parameter dominance(0.f); //constant allele dominance
	GO_Fish::do_nothing preserve; //don't preserve alleles from any generation
	GO_Fish::do_nothing sample_strategy; //only sample final generation

	GO_Fish::run_sim(a,mutation1,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy);
	std::cout<<std::endl<<"final number of mutations: " << a.num_mutations() << std::endl;

	a.sim_input_constants.num_generations = pow(10.f,3);//36;//50;//
	a.sim_input_constants.prev_sim_sample = 0;
	GO_Fish::const_parameter mutation2(pow(10.f,-9)); //per-site mutation rate
	GO_Fish::run_sim(a,mutation2,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy,a);
	std::cout<<std::endl<<"final number of mutations: " << a.num_mutations() << std::endl;

	a.sim_input_constants.init_mse = true;
	a.sim_input_constants.seed1 = 0xdecafbad; //random number seeds
	a.sim_input_constants.seed2 = 0xbeeff00d;
	a.sim_input_constants.num_generations = pow(10.f,3);//36;//50;//
	GO_Fish::run_sim(a,mutation2,demography,migration,selection,inbreeding,dominance,DFE,preserve,sample_strategy);
	std::cout<<std::endl<<"final number of mutations: " << a.num_mutations() << std::endl;
}

////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////

double gx(double x, double gamma, double mu_site, double L){
	if(gamma != 0) return 2*mu_site*L*(1-exp(-1*gamma*(1-x)))/((1-exp(-1*gamma))*x*(1-x));
	return 2*mu_site*L/x;
}


double* G(double gamma,double mu_site, double L, double N_chrome){
	double total_SNPs = 0;
	double* g = new double[(int)N_chrome];

	for(int j = 1; j <= (N_chrome - 1); j++){
		double freq = j/(N_chrome);
		g[j] = gx(freq, gamma, mu_site, L);
		total_SNPs += g[j];
	}

	g[0] = L-total_SNPs;

	return g;
}

void run_validation_test(){

	GO_Fish::allele_trajectories b;
    float gamma = 0; //effective selection
	float h = 0.5; //dominance
	float F = 0.0; //inbreeding
	int N_ind = 0.03*pow(10.f,5)*(1+F);//300;// //bug at N_ind = 300, F =0.0, gamma = 0//number of individuals in population, set to maintain consistent effective number of chromosomes
	float s = gamma/(2*N_ind); //selection coefficient
	float mu = pow(10.f,-9); //per-site mutation rate
	int total_number_of_generations = pow(10.f,3);//0;//1000;//1;//36;//
	b.sim_input_constants.num_sites = 10*2*pow(10.f,7); //number of sites
	float m = 0.00; //migration rate
	b.sim_input_constants.num_populations = 1; //number of populations
	int num_iter = 50;
    bool DFE = false;
    b.sim_input_constants.compact_rate = 35;
    double* expectation = G(gamma,mu, b.sim_input_constants.num_sites, 2.0*N_ind/(1.0+F));
    double expected_total_SNPs = b.sim_input_constants.num_sites-expectation[0];

	for(int i = 0; i < num_iter; i++){
		b.sim_input_constants.seed1 = 0xbeeff00d + 2*i; //random number seeds
		b.sim_input_constants.seed2 = 0xdecafbad - 2*i;
		GO_Fish::run_sim(b, GO_Fish::const_parameter(mu), GO_Fish::const_demography(N_ind), GO_Fish::const_equal_migration(m,b.sim_input_constants.num_populations), GO_Fish::const_selection(s), GO_Fish::const_parameter(F), GO_Fish::const_parameter(h), DFE, GO_Fish::do_nothing(), GO_Fish::do_nothing(), GO_Fish::time_sample());
		if(i==0){ std::cout<< "chi-gram number of mutations:"<<std::endl; }
		std::cout<< (int)expected_total_SNPs << "\t" << b[0]->num_mutations<< "\t" << ((b[0]->num_mutations - expected_total_SNPs)/expected_total_SNPs) << std::endl;
	}

	delete [] expectation;
}
